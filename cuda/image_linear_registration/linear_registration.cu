#include "TIPL/tipl.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


int main(void)
{
    // 1: load example image
    tipl::image<3> hfrom;
    if(!hfrom.load_from_file<tipl::io::nifti>("100206_T1w.nii"))
    {
        std::cout << "cannot find the sample file" << std::endl;
        return 1;
    }
    tipl::downsample_with_padding(hfrom);
    tipl::downsample_with_padding(hfrom);
    tipl::vector<3> voxel_size(1.0f,1.0f,1.0f);


    // 2: setup the transformation for linear registration algorithms to solve

    tipl::affine_transform<float> affine = {15.0,0,0,0.1f,0.02f,0,1.1f,1.2f,0.95f,0.02f,0,0};
    std::cout << "ground truth=\n" << affine;

    // 3: get the transformed image
    tipl::image<3> hto(hfrom.shape());
    tipl::transformation_matrix<float> trans0(tipl::affine_transform<float>(),
                                              hfrom.shape(),voxel_size,hto.shape(),voxel_size);
    tipl::transformation_matrix<float> trans(affine,hfrom.shape(),voxel_size,hto.shape(),voxel_size);
    tipl::resample_mt(hfrom,hto,trans);

    // 4: now use transformed image to calculate the transformation
    bool terminated = false;
    {
        std::cout << "\ncost function using cpu" << std::endl;
        tipl::reg::mutual_information mi;
        {
            tipl::time t("cpu time for cost function:");
            for(unsigned int i = 0;i < 20;++i)
                mi(hfrom,hto,trans0);
            std::cout << "cpu result:" << mi(hfrom,hto,trans0) << std::endl;
        }
        std::cout << "\ncost function using gpu" << std::endl;
        tipl::reg::mutual_information_cuda mi2;
        {
            tipl::time t("gpu time for cost function:");
            for(unsigned int i = 0;i < 20;++i)
                mi2(hfrom,hto,trans0);
            std::cout << "gpu result:" << mi2(hfrom,hto,trans0) << std::endl;
        }

    }

    {
        std::cout << "\nsolve using cpu" << std::endl;
        tipl::time t("cpu time (ms):");
        tipl::affine_transform<float> answer;
        tipl::reg::linear_two_way<tipl::reg::mutual_information> // use cpu multithread to calculate the cost function
                (hto,voxel_size,hfrom,voxel_size,answer,tipl::reg::affine,[&](void){return terminated;});
        std::cout << "cpu answer:\n" << answer;
    }

    {
        std::cout << "\nsolve using gpu" << std::endl;
        tipl::time t("gpu time (ms):");
        tipl::affine_transform<float> answer;
        tipl::reg::linear_two_way<tipl::reg::mutual_information_cuda> // use cuda to calculate the cost function
                (hto,voxel_size,hfrom,voxel_size,answer,tipl::reg::affine,[&](void){return terminated;});
        std::cout << "gpu answer:\n" << answer;
        std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
    }
    return 0;
}
